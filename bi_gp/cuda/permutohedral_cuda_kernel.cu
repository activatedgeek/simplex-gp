#include "hip/hip_runtime.h"
#include <algorithm>
#include <assert.h>
#include <cmath>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <c10/cuda/CUDACachingAllocator.h>
#include <torch/extension.h>
#include <THC/THCAtomics.cuh>

using at::Tensor;
typedef std::chrono::high_resolution_clock Clock;

// #define DEBUG

#define NANO_CAST(d) std::chrono::duration_cast<std::chrono::nanoseconds>(d)
#define BLOCK_SIZE 1024
#define PTAccessor(T,dim) at::PackedTensorAccessor32<T,dim,at::RestrictPtrTraits>
#define Ten2PTAccessor(T,x,dim) x.packed_accessor32<T,dim,at::RestrictPtrTraits>()
#define TenSize2D(m,n) {static_cast<int64_t>(m), static_cast<int64_t>(n)}
#define TenOptType(T, D) torch::dtype(T).device(D.type(),D.index())

// https://stackoverflow.com/a/14038590/2425365
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

template <typename scalar_t>
struct ReplayEntry{
  size_t entry;
  scalar_t weight;
};

template <typename scalar_t>
class HashTableGPU {
private:
  int16_t* keys;
  scalar_t* values;
  scalar_t* bufferValues;

  /**
   * Each point has at most (pd + 1) neighbors.
   * Each entry then maps to the lattice point.
   **/
  int* entry2nid;
  uint8_t* uqentry;
public:
  size_t N, pd, vd, capacity;
  #ifdef DEBUG
  int64_t* M; // for actual size after splat.
  #endif

  HashTableGPU(size_t pd_, size_t vd_, size_t N_): 
    pd(pd_), vd(vd_), N(N_) {
    capacity = N * (pd + 1);

    gpuErrchk(hipMallocManaged(&keys, capacity * pd * sizeof(int16_t)));

    gpuErrchk(hipMallocManaged(&values, capacity * vd * sizeof(scalar_t)));
    gpuErrchk(hipMallocManaged(&bufferValues, capacity * vd * sizeof(scalar_t)));
    for (size_t i = 0; i < capacity * vd; ++i) {
      values[i] = static_cast<scalar_t>(0.0);
      bufferValues[i] = static_cast<scalar_t>(0.0);
    }

    gpuErrchk(hipMallocManaged(&entry2nid, capacity * sizeof(int)));
    gpuErrchk(hipMallocManaged(&uqentry, capacity * sizeof(uint8_t)));
    for (size_t i = 0; i < capacity; ++i) {
      entry2nid[i] = static_cast<int>(-1);
      uqentry[i] = static_cast<uint8_t>(0);
    }

    #ifdef DEBUG
    gpuErrchk(hipMallocManaged(&M, sizeof(int64_t)));
    *M = static_cast<int64_t>(0);
    #endif
  }

  /**
   * NOTE: Copy constructor in kernel arguments triggers the destructor.
   * Instead, manually free this in the PermutohedralLatticeGPU class.
   **/
  void free() {
    gpuErrchk(hipFree(keys));
    gpuErrchk(hipFree(values));
    gpuErrchk(hipFree(bufferValues));
    gpuErrchk(hipFree(entry2nid));
    gpuErrchk(hipFree(uqentry));
  }

  __device__ __forceinline__ int* getEntries() { return entry2nid; }

  __device__ __forceinline__ uint8_t* getUqEntries() { return uqentry; }

  __device__ __forceinline__ int16_t* getKey(const size_t h) {
    if (entry2nid[h] == -1) {
      return nullptr;
    }
    return &keys[entry2nid[h] * pd];
  }

  __device__ __forceinline__ scalar_t* getValue(const size_t h) {
    if (entry2nid[h] == -1) {
      return nullptr;
    }
    return &values[entry2nid[h] * vd];
  }

  __device__ __forceinline__ scalar_t* getBufferValue(const size_t h) {
    if (entry2nid[h] == -1) {
      return nullptr;
    }
    return &bufferValues[entry2nid[h] * vd];
  }

  void swapBuffer() {
    scalar_t* tmp = values;
    values = bufferValues;
    bufferValues = tmp;
    for (size_t i = 0; i < capacity * vd; ++i) {
      bufferValues[i] = static_cast<scalar_t>(0.0);
    }
  }

  __device__ __forceinline__ size_t modhash(const int16_t* key) {
    size_t k = 0;
    for (uint16_t i = 0; i < pd; ++i) {
      k += static_cast<size_t>(key[i]);
      k *= static_cast<size_t>(2531011);
    }
    return k % capacity;
  }

  __device__ size_t insert(const int16_t* key, const int nid) {
    size_t h = modhash(key);

    while (true) {
      int cas = atomicCAS(&entry2nid[h], -1, -2); // Returns the (old) value at location.

      if (cas == -2) { // Locked by another thread.
      } else if (cas == -1) { // Lock acquired.
        for (size_t i = 0; i < pd; ++i) {
          keys[nid * pd + i] = key[i];
        }

        atomicExch(&entry2nid[h], nid);

        return h;
      } else { // Otherwise check if an existing key matches.
        bool match = true;
        for (size_t i = 0; i < pd && match; ++i) {
          match = keys[cas * pd + i] == key[i];
        }
        if (match) {
          return h;
        }
      }

      // Linear probing.
      ++h;
      if (h == capacity) {
        h = 0;
      }
    }
  }

  __device__ int get(const int16_t* key) {
    size_t h = modhash(key);
    bool loop = false;

    while (true) {
      int nid = entry2nid[h];
      if (nid == -1) {
        return -1;
      }

      bool match = true;
      for (size_t i = 0; i < pd && match; ++i) {
        match = keys[nid * pd + i] == key[i];
      }
      if (match) {
        return h;
      }

      ++h;
      if (h == capacity) {
        h = 0;
        // Linear probe finished.
        if (loop) {
          return -1;
        }
        loop = true;
      }
    }
  }
};

template <typename scalar_t>
__global__ void splat_kernel(
    const PTAccessor(scalar_t,2) ref,
    PTAccessor(scalar_t,2) matE,
    PTAccessor(int16_t,2) matY,
    PTAccessor(int16_t,2) matR,
    PTAccessor(scalar_t,2) matB,
    int16_t* matK,
    const scalar_t* scaleFactor,
    const int16_t* canonical,
    HashTableGPU<scalar_t> table,
    ReplayEntry<scalar_t>* replay) {
  const size_t n = blockIdx.x * blockDim.x + threadIdx.x;
  if (n >= ref.size(0)) {
    return;
  }

  const uint16_t pd = ref.size(1);
  auto pos = ref[n];
  auto elevated = matE[n];
  auto y = matY[n];
  auto rank = matR[n];
  auto bary = matB[n];
  auto key = &matK[n * pd];

  elevated[pd] = - pd * pos[pd - 1] * scaleFactor[pd - 1];
  for (uint16_t i = pd - 1; i > 0; i--) {
    elevated[i] = elevated[i + 1] - i * pos[i - 1] * scaleFactor[i - 1] +
                  (i + 2) * pos[i] * scaleFactor[i];
  }
  elevated[0] = elevated[1] + 2.0 * pos[0] * scaleFactor[0];

  int16_t h = 0;
  for (uint16_t i = 0; i <= pd; ++i) {
    y[i] = static_cast<int16_t>(round(elevated[i] / (pd + 1))) * (pd + 1);
    h += y[i];

    rank[i] = 0;
    bary[i] = 0.0;
  }
  h /= (pd + 1);

  bary[pd + 1] = 0.0;

  for (uint16_t i = 0; i < pd; ++i) {
    for (uint16_t j = i + 1; j <= pd; ++j) {
      if (elevated[i] - y[i] < elevated[j] - y[j]) {
        rank[i]++;
      } else {
        rank[j]++;
      }
    }
  }

  if (h > 0) {
    for (uint16_t i = 0; i <= pd; ++i) {
      if (rank[i] >= pd + 1 - h) {
          y[i] -= pd + 1;
          rank[i] += h - (pd + 1);
      }
      else {
        rank[i] += h;
      }
    }
  } else if (h < 0) {
    for (uint16_t i = 0; i <= pd; ++i) {
      if (rank[i] < -h) {
        y[i] += pd + 1;
        rank[i] += h + (pd + 1);
      } else {
        rank[i] += h;
      }
    }
  }

  for (uint16_t i = 0; i <= pd; ++i) {
    scalar_t delta = static_cast<scalar_t>(elevated[i] - y[i]) / (pd + 1);
    bary[pd - rank[i]] += delta;
    bary[pd + 1 - rank[i]] -= delta;
  }
  bary[0] += 1.0 + bary[pd + 1];

  for (uint16_t r = 0; r <= pd; ++r) {
    size_t nid = n * (pd + 1) + r;

    for (uint16_t i = 0; i < pd; ++i) {
      key[i] = y[i] + canonical[r * (pd + 1) + rank[i]];
    }

    replay[nid].entry = table.insert(key, nid);
    replay[nid].weight = bary[r];
  }
}

template <typename scalar_t>
__global__ void process_hashtable_kernel(
  HashTableGPU<scalar_t> table) {
  /**
   * NOTE: Hash table may have duplicate entries because
   * linear probing is not atomic. Assign every entry to
   * the first key match to correct this.
   **/  
  const size_t n = blockIdx.x * blockDim.x + threadIdx.x;
  if (n >= table.N) {
    return;
  }
  const size_t pd = table.pd;
  // const size_t r = blockIdx.y;
  int* entries = table.getEntries();
  uint8_t* uqentries = table.getUqEntries();
  
  for (size_t r = 0; r <= pd; ++r) {
    const size_t nid = n * (pd + 1) + r;

    if (entries[nid] >= 0) {
      auto h = table.get(table.getKey(nid));

      // Every element not re-assigned is unique.
      if (entries[nid] == entries[h]) {
        uqentries[nid] = static_cast<uint8_t>(1);
        #ifdef DEBUG
        gpuAtomicAdd(table.M, static_cast<int64_t>(1));
        #endif
      }

      entries[nid] = entries[h];
    }
  }
}

template <typename scalar_t>
__global__ void splat_value_kernel(
    const PTAccessor(scalar_t,2) src,
    HashTableGPU<scalar_t> table,
    ReplayEntry<scalar_t>* replay) {
  
  const size_t n = blockIdx.x * blockDim.x + threadIdx.x;
  if (n >= table.N) {
    return;
  }
  const size_t pd = table.pd;
  const size_t vd = src.size(1);
  // const size_t r = blockIdx.y;
  auto value = src[n];
  
  for (size_t r = 0; r <= pd; ++r) {
    const size_t nid = n * (pd + 1) + r;
    scalar_t* val = table.getValue(replay[nid].entry);
    for (size_t i = 0; i < vd; ++i) {
      gpuAtomicAdd(&val[i], replay[nid].weight * value[i]);
    }
  }
}

template <typename scalar_t>
__global__ void blur_kernel(
    HashTableGPU<scalar_t> table,
    const size_t ax,
    const PTAccessor(scalar_t,1) coeffs,
    int16_t* neighbors,
    const scalar_t* zero) {
  const size_t n = blockIdx.x * blockDim.x + threadIdx.x;
  if (n >= table.N) {
    return;
  }
  const size_t pd = table.pd;
  const size_t vd = table.vd;
  const uint8_t* uqentries = table.getUqEntries();
  int16_t* neighbor = &neighbors[n * (pd + 1)];

  for (size_t r = 0; r <= pd; ++r) {
    const size_t nid = n * (pd + 1) + r;
    if (!uqentries[nid]) {
      // Must have been processed at its lattice node.
      continue;
    }

    const int16_t* key = table.getKey(nid);
    scalar_t* bufferVal = table.getBufferValue(nid);
    const int16_t order = static_cast<int16_t>(coeffs.size(0) / 2);

    for (int16_t o = -order; o <= order; ++o) {
      for (size_t p = 0; p < pd; ++p) {
        neighbor[p] = key[p] - o;
      }
      neighbor[ax] = key[ax] + o * pd;

      int h = table.get(neighbor);
      const scalar_t* val = h >= 0 ? table.getValue(h) : zero;
      for (size_t v = 0; v < vd; ++v) {
        bufferVal[v] += coeffs[o + order] * val[v];
      }
    }
  }
}

template <typename scalar_t>
__global__ void slice_kernel(
    PTAccessor(scalar_t,2) res,
    HashTableGPU<scalar_t> table,
    ReplayEntry<scalar_t>* replay) {
  const size_t n = blockIdx.x * blockDim.x + threadIdx.x;
  if (n >= res.size(0)) {
    return;
  }

  const size_t pd = table.pd;
  const size_t vd = res.size(1);
  auto out = res[n];
  /** NOTE: magic? scaling constant. **/
  const scalar_t scale = 1.0 + pow(2.0, - (scalar_t) pd);

  for (size_t r = 0; r <= pd; ++r) {
    size_t nid = n * (pd + 1) + r;
    scalar_t* val = table.getValue(replay[nid].entry);
    for (size_t j = 0; j < vd; ++j) {
      out[j] += (replay[nid].weight * val[j]) / scale;
    }
  }
}

template <typename scalar_t>
class PermutohedralLatticeGPU {
private:
  uint16_t pd, vd;
  size_t N, order;
  scalar_t* scaleFactor;
  int16_t* canonical;
  HashTableGPU<scalar_t> hashTable;
  ReplayEntry<scalar_t>* replay;
public:
  PermutohedralLatticeGPU(uint16_t pd_, uint16_t vd_, size_t N_, const scalar_t filter_var): 
    pd(pd_), vd(vd_), N(N_), hashTable(HashTableGPU<scalar_t>(pd_, vd_, N_)) {
    
    scalar_t invStdDev = (pd + 1) * sqrt(filter_var +  static_cast<scalar_t>(1.0 / 6.0));

    gpuErrchk(hipMallocManaged(&scaleFactor, pd * sizeof(scalar_t)));
    for (uint16_t i = 0; i < pd; ++i) {
      scaleFactor[i] = invStdDev / static_cast<scalar_t>(sqrt((i + 1) * (i + 2)));
    }

    gpuErrchk(hipMallocManaged(&canonical, (pd + 1) * (pd + 1) * sizeof(int16_t)));
    for (uint16_t i = 0; i <= pd; ++i) {
      for (uint16_t j = 0; j <= pd - i; ++j) {
        canonical[i * (pd + 1) + j] = i;
      }
      for (uint16_t j = pd - i + 1; j <= pd; ++j) {
        canonical[i * (pd + 1) + j] = i - (pd + 1);
      }
    }

    gpuErrchk(hipMallocManaged(&replay, N * (pd + 1) * sizeof(ReplayEntry<scalar_t>)));
  }

  ~PermutohedralLatticeGPU() {
    hashTable.free();
    gpuErrchk(hipFree(scaleFactor));
    gpuErrchk(hipFree(canonical));
    gpuErrchk(hipFree(replay));
  }

  void splat(Tensor src, Tensor ref) {
    Tensor _matE = torch::zeros(TenSize2D(N, pd + 1), TenOptType(ref.dtype(),ref.device()));
    Tensor _matY = torch::zeros(TenSize2D(N, pd + 1), TenOptType(torch::kI16,ref.device()));
    Tensor _matR = torch::zeros(TenSize2D(N, pd + 1), TenOptType(torch::kI16,ref.device()));
    Tensor _matB = torch::zeros(TenSize2D(N, pd + 2), TenOptType(ref.dtype(),ref.device()));
    int16_t* _matK = nullptr;
    gpuErrchk(hipMallocManaged(&_matK, N * pd * sizeof(int16_t)));

    const dim3 threads(BLOCK_SIZE);
    const dim3 blocks((N + threads.x - 1) / threads.x);

    splat_kernel<scalar_t><<<blocks, threads>>>(
      Ten2PTAccessor(scalar_t,ref,2),
      Ten2PTAccessor(scalar_t,_matE,2), Ten2PTAccessor(int16_t,_matY,2),
      Ten2PTAccessor(int16_t,_matR,2), Ten2PTAccessor(scalar_t,_matB,2),
      _matK,
      scaleFactor, canonical,
      hashTable, replay);
    gpuErrchk(hipPeekAtLastError());

    // const dim3 proc_blocks((N + threads.x - 1) / threads.x, pd + 1);

    process_hashtable_kernel<scalar_t><<<blocks,threads>>>(hashTable);
    gpuErrchk(hipPeekAtLastError());

    splat_value_kernel<scalar_t><<<blocks,threads>>>(
      Ten2PTAccessor(scalar_t,src,2),
      hashTable, replay);
    gpuErrchk(hipPeekAtLastError());

    gpuErrchk(hipFree(_matK));
  }

  void blur(const Tensor coeffs) {
    int16_t* _matNeK;
    gpuErrchk(hipMallocManaged(&_matNeK, N * (pd + 1) * sizeof(int16_t)));

    scalar_t* zero;
    gpuErrchk(hipMallocManaged(&zero, vd * sizeof(scalar_t)));
    for (size_t d = 0; d < vd; ++d) {
      zero[d] = static_cast<scalar_t>(0.0);
    }

    std::vector<size_t> dims;
    for (size_t d = 0; d <= pd; ++d) {
      dims.push_back(d);
    }
    /** TODO: **/
    // std::random_shuffle(dims.begin(), dims.end());

    const dim3 threads(BLOCK_SIZE);
    const dim3 blocks((N + threads.x - 1) / threads.x);

    for (size_t ax: dims) {
      blur_kernel<scalar_t><<<blocks, threads>>>(
        hashTable, ax, Ten2PTAccessor(scalar_t,coeffs,1),
        _matNeK, zero);
      gpuErrchk(hipPeekAtLastError());

      gpuErrchk(hipDeviceSynchronize());
      hashTable.swapBuffer();
    }

    gpuErrchk(hipFree(zero));
    gpuErrchk(hipFree(_matNeK));
  }

  Tensor slice(Tensor src, Tensor ref) {
    Tensor res = torch::zeros(TenSize2D(N, vd), TenOptType(src.dtype(),src.device()));

    const dim3 threads(BLOCK_SIZE);
    const dim3 blocks((N + threads.x - 1) / threads.x);

    slice_kernel<scalar_t><<<blocks, threads>>>(
      Ten2PTAccessor(scalar_t,res,2),
      hashTable, replay);
    gpuErrchk(hipPeekAtLastError());

    gpuErrchk(hipDeviceSynchronize());
    return res;
  }

  Tensor filter(Tensor src, Tensor ref, const Tensor coeffs) {
    #ifdef DEBUG
    auto start_ts = Clock::now();
    #endif

    splat(src, ref);

    #ifdef DEBUG
    auto elapsed_ts = NANO_CAST(Clock::now() - start_ts).count();
    std::cout << "Hash table size: " << *hashTable.M << std::endl;
    std::cout << "Splat: " << elapsed_ts << " ns" << std::endl;
    #endif

    c10::cuda::CUDACachingAllocator::emptyCache();

    #ifdef DEBUG
    start_ts = Clock::now();
    #endif

    blur(coeffs);

    #ifdef DEBUG
    elapsed_ts = NANO_CAST(Clock::now() - start_ts).count();
    std::cout << "Blur: " << elapsed_ts << " ns" << std::endl;
    #endif

    #ifdef DEBUG
    start_ts = Clock::now();
    #endif

    auto res = slice(src, ref);

    #ifdef DEBUG
    elapsed_ts = NANO_CAST(Clock::now() - start_ts).count();
    std::cout << "Slice: " << elapsed_ts << " ns" << std::endl;
    #endif

    return res;
  }
};

template <typename scalar_t>
scalar_t variance(Tensor coeffs){
  auto coeffs_iter = coeffs.accessor<scalar_t,1>();

  scalar_t mom0 = static_cast<scalar_t>(0.0);
  scalar_t mom1 = static_cast<scalar_t>(0.0);
  scalar_t mom2 = static_cast<scalar_t>(0.0);

  int16_t k = static_cast<int16_t>(coeffs.size(0));
  for (int16_t i = 0; i < k; ++i) {
    scalar_t c = coeffs_iter[i];
    mom0 += c;
    mom1 += i * c;
    mom2 += i * i * c;
  }
  scalar_t mean = mom1 / mom0;
  scalar_t var = mom2 / mom0 - mean * mean;
  return var;
}

Tensor permutohedral_cuda_filter(Tensor src, Tensor ref, const Tensor coeffs) {
  Tensor out;

  AT_DISPATCH_FLOATING_TYPES(src.scalar_type(), "permutohedral_lattice", ([&]{
    const Tensor cpu_coeffs = coeffs.to(torch::kCPU, false);
    const scalar_t filter_var = variance<scalar_t>(cpu_coeffs);
    PermutohedralLatticeGPU<scalar_t> lattice(ref.size(-1), src.size(-1),
                                              src.size(0), filter_var);
    out = lattice.filter(src, ref, coeffs);
  }));

  return out;
}
